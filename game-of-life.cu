/** Author: alexge50
 *  How to use: input should be given in a file input.txt, in the same directory as the binary. Output is given in output.txt
 *  Input:  [Number of steps]
 *          [height - number of rows] [width - number of columns]
 *          board
 *  Output: [time] ms
 *          board at the current state when execution was stopped
 *  Compilation requires no other option than default:
 *     nvcc game-of-life.cout
 *     ./a.out
 **/


#include <hip/hip_runtime.h>
#include <stdio.h>

#include <sys/time.h>

inline long long GetTime()
{
    struct timeval tv;

    gettimeofday(&tv, NULL);
    return tv.tv_sec * 1000000LL + tv.tv_usec;
}

__global__ void update(int *board, int *result_board, int nRows, int nColumns)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int j = threadIdx.x;
  int i = blockIdx.x;

  if(i >= 1 && i <= nRows - 2 && j >= 1 && j <= nColumns - 2)
  {
    int n_neighbors = 0;
    /*for(int di = -1; di <= 1; di++)
    {
      for(int dj = -1; dj <= 1; dj++)
      {
        int _index = (i + di) * nColumns + (j + dj);
        n_neighbors += board[_index] && !(di == 0 && dj == 0);
      }
    }*/
    n_neighbors = board[(i - 1) * nColumns + (j + 1)] +
                  board[(i + 1) * nColumns + (j + 1)] +
                  board[(i + 1) * nColumns + (j - 1)] +
                  board[(i + 0) * nColumns + (j + 1)] +
                  board[(i + 0) * nColumns + (j - 1)] +
                  board[(i + 1) * nColumns + (j + 0)] +
                  board[(i - 1) * nColumns + (j + 0)] +
                  board[(i - 1) * nColumns + (j - 1)];

    if(board[index])
      atomicExch(&result_board[index], n_neighbors == 2 || n_neighbors == 3);
    else atomicExch(&result_board[index], n_neighbors == 3);
  }
}

int main()
{
    FILE *fin = fopen("input.txt", "r");
    FILE *fout = fopen("output.txt", "w");
    int nSteps;
    int nRows, nColumns;
    long long timeStart, timeStop;

    int *board;
    int *device_board0, *device_board1;
    //char *device_board[2];

    fscanf(fin, "%d %d %d ", &nSteps, &nRows, &nColumns);
    //cudaMalloc((void **) &device_board[0], sizeof(char) * (nRows + 2) * (nColumns + 2));
    //cudaMalloc((void **) &device_board[1], sizeof(char) * (nRows + 2) * (nColumns + 2));

    nColumns += 2;
    nRows += 2;

    board = (int*)malloc(sizeof(int) * (nRows) * (nColumns));

    hipMalloc((void **) &device_board0, sizeof(int) * (nRows) * (nColumns));
    hipMalloc((void **) &device_board1, sizeof(int) * (nRows) * (nColumns));


    for(int i = 0; i < nRows; i++)
      for(int j = 0; j < nColumns; j++)
        board[i * nColumns + j] = 0;

    for(int i = 1; i <= nRows - 2; ++i)
        for (int j = 1; j <= nColumns - 2; ++j)
        {
            char cell;
            fscanf(fin, "%c ", &cell);
            board[i * nColumns + j] = (cell == '*');
        }

    hipMemcpy(device_board0, board, sizeof(int) * (nRows) * (nColumns), hipMemcpyHostToDevice);
    hipMemcpy(device_board1, board, sizeof(int) * (nRows) * (nColumns), hipMemcpyHostToDevice);

    timeStart = GetTime();
    int i = 0;
    for (int k = 0; k < nSteps; ++k)
    {
      //UpdateCall
      //update<<<nRows, nColumns>>>(device_board[i], device_board[i - 1], nRows, nColumns);
      if(i == 0)
    	  update<<<nRows, nColumns>>>(device_board0, device_board1, nRows, nColumns);
      else
    	  update<<<nRows, nColumns>>>(device_board1, device_board0, nRows, nColumns);
      i = 1 - i;
    }
    //cudaMemcpy(board, device_board[i], sizeof(char) * (nRows + 2) * (nColumns + 2), cudaMemcpyDeviceToHost);
    if(i == 0)
    	hipMemcpy(board, device_board0, sizeof(int) * (nRows) * (nColumns), hipMemcpyDeviceToHost);
    else hipMemcpy(board, device_board1, sizeof(int) * (nRows) * (nColumns), hipMemcpyDeviceToHost);
    timeStop = GetTime();

    long double deltaTime = static_cast<long double>(timeStop - timeStart) / static_cast<long double>(1000.); // microseconds to milli seconds

    fprintf(fout, "[%Lf ms]\n", deltaTime);

    for(int i = 1; i <= nRows - 2; ++i)
    {
        for(int j = 1; j <= nColumns - 2; ++j)
            fprintf(fout, "%c", board[i * nColumns + j] ? '*' : '.');
        fprintf(fout, "\n");
    }

    printf("Time: %Lf ms\n", deltaTime);

    free(board);
    hipFree(device_board0);
    hipFree(device_board1);

    fclose(fin);
    fclose(fout);
    return 0;
}
